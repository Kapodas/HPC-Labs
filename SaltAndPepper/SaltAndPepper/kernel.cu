#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <texture_indirect_functions.h>
#include "EasyBMP.h"

#define BLOCK_SIZE 16

__global__ void medianFilter(unsigned char* output, int width, int height, hipTextureObject_t texObj);

int main() {
    BMP InputImage;
    InputImage.ReadFromFile("Lena.bmp");

    int width = InputImage.TellWidth();
    int height = InputImage.TellHeight();

    unsigned char* h_input = new unsigned char[width * height];
    unsigned char* h_output = new unsigned char[width * height];

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            h_input[j * width + i] = InputImage(i, j)->Red; // Assuming grayscale image
        }
    }

    unsigned char* d_input;
    unsigned char* d_output;
    hipMalloc(&d_input, width * height * sizeof(unsigned char));
    hipMalloc(&d_output, width * height * sizeof(unsigned char));

    hipMemcpy(d_input, h_input, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Bind texture memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    hipMemcpyToArray(cuArray, 0, 0, h_input, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

    // Launch kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    medianFilter << <dimGrid, dimBlock >> > (d_output, width, height, texObj);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Processing time: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_output, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save output image
    BMP OutputImage;
    OutputImage.SetSize(width, height);
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            RGBApixel pixel;
            pixel.Red = h_output[j * width + i];
            pixel.Green = h_output[j * width + i];
            pixel.Blue = h_output[j * width + i];
            pixel.Alpha = 0;
            OutputImage.SetPixel(i, j, pixel);
        }
    }
    OutputImage.WriteToFile("output.bmp");

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFreeArray(cuArray);
    hipDestroyTextureObject(texObj);
    delete[] h_input;
    delete[] h_output;

    return 0;
}

__global__ void medianFilter(unsigned char* output, int width, int height, hipTextureObject_t texObj) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    unsigned char window[9];
    int idx = 0;

    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int nx = x + dx;
            int ny = y + dy;
            if (nx < 0) nx = 0;
            if (ny < 0) ny = 0;
            if (nx >= width) nx = width - 1;
            if (ny >= height) ny = height - 1;
            window[idx++] = tex2D<unsigned char>(texObj, nx, ny);
        }
    }

    // Sort the window array (using bubble sort for simplicity)
    for (int i = 0; i < 9; i++) {
        for (int j = i + 1; j < 9; j++) {
            if (window[i] > window[j]) {
                unsigned char temp = window[i];
                window[i] = window[j];
                window[j] = temp;
            }
        }
    }

    // Median is the middle element
    output[y * width + x] = window[4];
}