#include "hip/hip_runtime.h"
﻿#include "includes.cuh"

using namespace std;

hipError_t addWithCuda(vector<double>& c, const vector<double>& a, const vector<double>& b, const size_t size);

__global__ void addKernel(double* c, const double* a, const double* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    fillVector fv;
    VectorSum vecSum;
    size_t size;
    cin >> size;

    vector<double> vecA(size);
    vector<double> vecB(size);
    vector<double> vecC;
    vector<double> vecD;
    fv.fillRandom(vecA);
    fv.fillRandom(vecB);
    auto start = chrono::high_resolution_clock::now();
    hipError_t cudaStatus = addWithCuda(vecD, vecA, vecB, size);
    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> elapsed = end - start;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    cout << "GPU sum execution time: " << elapsed.count() << " ms" << endl;
    try {
        start = chrono::high_resolution_clock::now();
        vecC = vecSum.cpuSum(vecA, vecB);
        end = chrono::high_resolution_clock::now();
        elapsed = end - start;
    }
    catch (const invalid_argument& e) {
        cerr << e.what();
    }
    cout << "CPU sum execution time: " << elapsed.count() << " ms" << endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t addWithCuda(vector<double>& c, const vector<double>& a, const vector<double>& b, const size_t size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a.data(), size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b.data(), size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    int blockSize = 1024; // Максимальное количество потоков в блоке
    int numBlocks = (size + blockSize - 1) / blockSize;
    addKernel <<< numBlocks, blockSize >>> (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    c.resize(size);
    cudaStatus = hipMemcpy(c.data(), dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
